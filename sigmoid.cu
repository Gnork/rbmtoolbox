
#include <hip/hip_runtime.h>
extern "C"
__global__
void sigmoid(float* a, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		a[i] = 1.0f / (expf(-a[i]) + 1.0f);
	}
}