
#include <hip/hip_runtime.h>
extern "C"

__global__
void contrastiveDivergence(float* positive, float* negative, float* weights, float learningRate, int n) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		weights[i] = weights[i] + (positive[i] - negative[i]) * learningRate;
	}
}